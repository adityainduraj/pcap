#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define WIDTH 10
#define MASK_WIDTH 3

__global__ void convolution1D(int *N, int *M, int *P, int width, int mask_width) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int radius = mask_width / 2;
    int sum = 0;
    
    if (i < width) {
        for (int j = -radius; j <= radius; j++) {
            int index = i + j;
            if (index >= 0 && index < width) {
                sum += N[index] * M[j + radius];
            }
        }
        P[i] = sum;
    }
}

void printArray(int *array, int size) {
    for (int i = 0; i < size; i++) {
        printf("%d ", array[i]);
    }
    printf("\n");
}

int main() {
    int h_N[WIDTH] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
    int h_M[MASK_WIDTH] = {1, 0, -1};
    int h_P[WIDTH];
    
    int *d_N, *d_M, *d_P;
    size_t size_N = WIDTH * sizeof(int);
    size_t size_M = MASK_WIDTH * sizeof(int);
    
    hipMalloc((void**)&d_N, size_N);
    hipMalloc((void**)&d_M, size_M);
    hipMalloc((void**)&d_P, size_N);
    
    hipMemcpy(d_N, h_N, size_N, hipMemcpyHostToDevice);
    hipMemcpy(d_M, h_M, size_M, hipMemcpyHostToDevice);
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (WIDTH + threadsPerBlock - 1) / threadsPerBlock;
    
    convolution1D<<<blocksPerGrid, threadsPerBlock>>>(d_N, d_M, d_P, WIDTH, MASK_WIDTH);
    
    hipMemcpy(h_P, d_P, size_N, hipMemcpyDeviceToHost);
    
    printf("Resultant Array:\n");
    printArray(h_P, WIDTH);
    
    hipFree(d_N);
    hipFree(d_M);
    hipFree(d_P);
    
    return 0;
}

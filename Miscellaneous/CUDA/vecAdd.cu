#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define n 1024  // Vector size

__global__ void vecAddKernel(float* A, float* B, float* C, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i < n)
        C[i] = A[i] + B[i];
}

int main(void) {
    float *A, *B, *C;           // Host arrays
    float *d_A, *d_B, *d_C;     // Device arrays
    int size = n * sizeof(float);

    // Allocate memory on host
    A = (float*)malloc(size);
    B = (float*)malloc(size);
    C = (float*)malloc(size);

    // Initialize A and B
    for (int i = 0; i < n; i++) {
        A[i] = i * 1.0f;
        B[i] = (n - i) * 1.0f;
    }

    // Part-1: Allocate memory on device
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Part-2: Launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (int)ceil((float)n / threadsPerBlock);
    vecAddKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, n);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }

    // Part-3: Copy result back to host
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Print part of result for verification
    for (int i = 0; i < 10; i++) {
        printf("C[%d] = %f\n", i, C[i]);
    }

    // Free memory
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(A); free(B); free(C);

    return 0;
}

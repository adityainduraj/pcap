#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <conio.h>

#define N 1024 

__global__ void CUDACount(char* A, unsigned int *d_count, int len) { 
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < len && A[i] == 'a') { 
        atomicAdd(d_count, 1);
    }
}

int main() { 
    char A[N]; 
    char *d_A; 
    unsigned int count = 0; 
    unsigned int *d_count; 
    unsigned int result = 0;

    printf("Enter a string: "); 
    fgets(A, N, stdin);
    int len = strlen(A);

    // CUDA timing
    hipEvent_t start, stop; 
    hipEventCreate(&start); 
    hipEventCreate(&stop); 
    hipEventRecord(start, 0); 

    // Allocate memory on device
    hipMalloc((void**)&d_A, len * sizeof(char));
    hipMalloc((void**)&d_count, sizeof(unsigned int)); 

    // Copy data to device
    hipMemcpy(d_A, A, len * sizeof(char), hipMemcpyHostToDevice); 
    hipMemcpy(d_count, &count, sizeof(unsigned int), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (len + threadsPerBlock - 1) / threadsPerBlock;
    CUDACount<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_count, len); 

    hipError_t error = hipGetLastError(); 
    if (error != hipSuccess) { 
        printf("CUDA Error: %s\n", hipGetErrorString(error)); 
        return 1;
    } 

    hipEventRecord(stop, 0); 
    hipEventSynchronize(stop); 

    // Measure time
    float elapsedTime; 
    hipEventElapsedTime(&elapsedTime, start, stop); 

    // Copy result back to host
    hipMemcpy(&result, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost); 

    printf("Total occurrences of 'a' = %u\n", result); 
    printf("Time Taken = %f ms\n", elapsedTime); 

    // Free memory
    hipFree(d_A); 
    hipFree(d_count); 

    getch(); 
    return 0; 
}

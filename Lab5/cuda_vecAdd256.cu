#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void vecAdd(int* A, int* B, int* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        C[i] = A[i]+B[i];
    }
}

int main() {
    int N = 1024;
    size_t size = N * sizeof(int);

    int *h_A = (int*)malloc(size);
    int *h_B = (int*)malloc(size);
    int *h_C = (int*)malloc(size);
    
    for (int i = 0; i < N; i++) {
        h_A[i] = i;
        h_B[i] = 2 * i;
    }

    int *d_A, *d_B, *d_C;

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int numBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    vecAdd<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    printf("Result Vector C:\n");
    for (int i = 0; i < N; i++) {
        printf("%d + %d = %d\n", h_A[i], h_B[i], h_C[i]);
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}


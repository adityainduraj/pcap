#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void vectorAdd_blockSizeAsN(int* A, int* B, int* C, int N) {
    int index = threadIdx.x;
    if (index < N) {
        C[index] = A[index] + B[index];
    }
}

__global__ void vectorAdd_NThreads(int* A, int* B, int* C, int N) {
    int index = blockIdx.x;
    if (index < N) {
        C[index] = A[index] + B[index];
    }
}

int main() {
    int N = 10;
    size_t size = N * sizeof(int);

    int *h_A = (int*)malloc(size);
    int *h_B = (int*)malloc(size);
    int *h_C = (int*)malloc(size);
    
    for (int i = 0; i < N; i++) {
        h_A[i] = i;
        h_B[i] = 2 * i;
    }

    int *d_A, *d_B, *d_C;

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    vectorAdd_blockSizeAsN<<<1, N>>>(d_A, d_B, d_C, N);

    vectorAdd_NThreads<<<N, 1>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    printf("Result Vector C:\n");
    for (int i = 0; i < N; i++) {
        printf("%d + %d = %d\n", h_A[i], h_B[i], h_C[i]);
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}


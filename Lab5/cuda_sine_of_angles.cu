#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void calculateSine(float* input, float* output, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        output[i] = sinf(input[i]);
    }
}

int main() {
    int N = 10;
    size_t size = N * sizeof(float);

    float *h_input = (float*)malloc(size);
    float *h_output = (float*)malloc(size);

    for (int i = 0; i < N; i++) {
        h_input[i] = i * 0.1;
    }

    float *d_input, *d_output;
    hipMalloc(&d_input, size);
    hipMalloc(&d_output, size);

    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);
    int threadsPerBlock = 256;
    int numBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    calculateSine<<<numBlocks, threadsPerBlock>>>(d_input, d_output, N);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel launch failed: %s\n", hipGetErrorString(err));
        return -1;
    }
    hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);
    printf("Angle(rad):\tSines");
    for (int i = 0; i < N; i++) {
        printf("\nsin(%f):\t%f\n", h_input[i], h_output[i]);
    }
    hipFree(d_input);
    hipFree(d_output);
    free(h_input);
    free(h_output);
    return 0;
}


#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define MAX_WORDS 100
#define MAX_WORD_LENGTH 20

__device__ bool isWordMatch(const char *sentence, int start, const char *word, int wordLen, int sentenceLen) {
    if (start + wordLen > sentenceLen) return false;
    for (int i = 0; i < wordLen; i++) {
        if (sentence[start + i] != word[i]) return false;
    }
    return (start + wordLen == sentenceLen || sentence[start + wordLen] == ' ');
}

__global__ void countWordOccurrences(char *sentence, char *word, int *count, int sentenceLen, int wordLen) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < sentenceLen && (i == 0 || sentence[i - 1] == ' ')) {
        if (isWordMatch(sentence, i, word, wordLen, sentenceLen)) {
            atomicAdd(count, 1);
        }
    }
}

int main() {
    char h_sentence[] = "cuda is fast and cuda is powerful and cuda is parallel";
    char h_word[] = "cuda";
    int h_count = 0;

    char *d_sentence, *d_word;
    int *d_count;
    int sentenceLen = strlen(h_sentence);
    int wordLen = strlen(h_word);

    hipMalloc((void**)&d_sentence, sentenceLen + 1);
    hipMalloc((void**)&d_word, wordLen + 1);
    hipMalloc((void**)&d_count, sizeof(int));

    hipMemcpy(d_sentence, h_sentence, sentenceLen + 1, hipMemcpyHostToDevice);
    hipMemcpy(d_word, h_word, wordLen + 1, hipMemcpyHostToDevice);
    hipMemcpy(d_count, &h_count, sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (sentenceLen + threadsPerBlock - 1) / threadsPerBlock;
    countWordOccurrences<<<blocksPerGrid, threadsPerBlock>>>(d_sentence, d_word, d_count, sentenceLen, wordLen);

    hipMemcpy(&h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);

    printf("The word '%s' appears %d times in the sentence.\n", h_word, h_count);

    hipFree(d_sentence);
    hipFree(d_word);
    hipFree(d_count);

    return 0;
}

#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void calculateTotal(int* prices, int* quantities, int* total, int numItems) {
    int idx = threadIdx.x;
    if (idx < numItems) {
        atomicAdd(total, prices[idx] * quantities[idx]);
    }
}

int main() {
    int numItems;
    printf("Enter the number of items: ");
    scanf("%d", &numItems);

    int prices[numItems];
    int quantities[numItems];

    printf("Enter the prices of the items:\n");
    for (int i = 0; i < numItems; ++i) {
        scanf("%d", &prices[i]);
    }

    printf("Enter the quantities purchased by friends:\n");
    for (int i = 0; i < numItems; ++i) {
        scanf("%d", &quantities[i]);
    }

    int* d_prices;
    int* d_quantities;
    int* d_total;
    int total = 0;

    hipMalloc(&d_prices, numItems * sizeof(int));
    hipMalloc(&d_quantities, numItems * sizeof(int));
    hipMalloc(&d_total, sizeof(int));

    hipMemcpy(d_prices, prices, numItems * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_quantities, quantities, numItems * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_total, &total, sizeof(int), hipMemcpyHostToDevice);

    calculateTotal<<<1, numItems>>>(d_prices, d_quantities, d_total, numItems);

    hipMemcpy(&total, d_total, sizeof(int), hipMemcpyDeviceToHost);

    printf("Total purchase done by friends: %d\n", total);

    hipFree(d_prices);
    hipFree(d_quantities);
    hipFree(d_total);

    return 0;
}

